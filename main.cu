#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "TSPLIB_parser.h"
#include <hip/hip_runtime.h>

using namespace std;
#define dbg cout << __FILE__ << ":" << __LINE__ << ", " << endl

const int POP_SIZE = 100;
const int NUM_GEN = 1;
int m = POP_SIZE;
int n;
double **cost, **d_cost;
double *X, *Y, *d_X, *d_Y;
int *defaultArr;
int **initialPopulation;
int **pop1, **pop2, **ofsp;


void allocateCudaMemory() {
    double **temp = (double**) malloc(sizeof(double*)*n);
    for(int i = 0; i < n; i++) {
        hipMalloc(&temp[i], sizeof(double)*n);
        hipMemcpy(&temp[i], cost[i], sizeof(double)*n, hipMemcpyHostToDevice);
    }

    hipMalloc(&d_cost, sizeof(double*)*n);
    hipMemcpy(&d_cost, temp, sizeof(double*)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_X, sizeof(double)*n);
    hipMemcpy(&d_X, X, sizeof(double)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_Y, sizeof(double)*n);
    hipMemcpy(&d_Y, Y, sizeof(double)*n, hipMemcpyHostToDevice);
    
    return;
}

void makeInitialPopulation() {
    int **cpop1, **cpop2, **cofsp;
    cpop1 = new int*[POP_SIZE];
    cpop2 = new int*[POP_SIZE];
    cofsp = new int*[POP_SIZE];
    hipMallocManaged(&initialPopulation, sizeof(int*)*POP_SIZE);
    hipMalloc(&pop1, sizeof(int*)*POP_SIZE);
    hipMalloc(&pop2, sizeof(int*)*POP_SIZE);
    hipMalloc(&ofsp, sizeof(int*)*POP_SIZE);
    for(int i = 0; i < POP_SIZE; i++) {
        hipMallocManaged(&initialPopulation[i], sizeof(int)*n);
        hipMalloc(&cpop1[i], sizeof(int)*n);
        hipMalloc(&cpop2[i], sizeof(int)*n);
        hipMalloc(&cofsp[i], sizeof(int)*n);
        random_shuffle(defaultArr, defaultArr+n);
        for(int j = 0; j < n; j++) initialPopulation[i][j] = defaultArr[j];
    }
    // hipMemcpy(initialPopulation, cinitialPopulation, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(pop1, cpop1, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(pop2, cpop2, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(ofsp, cofsp, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    // for(int i = 0; i < POP_SIZE; i++) {
    //     for(int j = 0; j < n; j++) {
    //         cout << initialPopulation[i][j] << ",";
    //     }
    //     cout << endl;
    // }
    return;    
}

__global__ void copyKernel(int n, int POP_SIZE, int **pop1, int **pop2) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= POP_SIZE) 
        return;
    
    for(int i = 0; i < n; i++) 
        pop1[id][i] = pop2[id][i];

    return;
}

__global__ void processKernel(int n, int POP_SIZE, int **pop1, int **pop2) {
    
    return;
}

void runGA() {
    for(int genNum = 0; genNum < NUM_GEN; genNum++) {
        if(genNum == 0) 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, initialPopulation);        
        else 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, pop2);
        hipDeviceSynchronize();
    }
    return;
}

int main(int argc, char **argv) {
    string filename = "TSPLIB/";
    filename = filename + argv[1];
    cout << filename << endl;
    ReadFile(filename, n, cost, X, Y);

    allocateCudaMemory();

    defaultArr = new int[n];
    for(int i = 0; i < n; i++) 
        defaultArr[i] = i;
    
    makeInitialPopulation();

    runGA();
}

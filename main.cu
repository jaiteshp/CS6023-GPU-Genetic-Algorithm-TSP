#include <bits/stdc++.h>
#include "TSPLIB_parser.h"
#include <hip/hip_runtime.h>

using namespace std;
#define dbg cout << __FILE__ << ":" << __LINE__ << ", " << endl

const int POP_SIZE = 100;
int n;
double **cost, **d_cost;
double *X, *Y, *d_X, *d_Y;
int *defaultArr;
int **initialPopulation;


void allocateCudaMemory() {
    double **temp = (double**) malloc(sizeof(double*)*n);
    for(int i = 0; i < n; i++) {
        hipMalloc(&temp[i], sizeof(double)*n);
        hipMemcpy(&temp[i], cost[i], sizeof(double)*n, hipMemcpyHostToDevice);
    }

    hipMalloc(&d_cost, sizeof(double*)*n);
    hipMemcpy(&d_cost, temp, sizeof(double*)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_X, sizeof(double)*n);
    hipMemcpy(&d_X, X, sizeof(double)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_Y, sizeof(double)*n);
    hipMemcpy(&d_Y, Y, sizeof(double)*n, hipMemcpyHostToDevice);
    
    return;
}

void makeInitialPopulation() {
    hipMallocManaged(&initialPopulation, sizeof(int*)*POP_SIZE);
    for(int i = 0; i < POP_SIZE; i++) {
        hipMallocManaged(&initialPopulation[i], sizeof(int)*n);
        random_shuffle(defaultArr, defaultArr+n);
        for(int j = 0; j < n; j++) initialPopulation[i][j] = defaultArr[j];
    }
    for(int i = 0; i < POP_SIZE; i++) {
        for(int j = 0; j < n; j++) {
            cout << initialPopulation[i][j] << ",";
        }
        cout << endl;
    }
    return;    
}

int main(int argc, char **argv) {
    string filename = "TSPLIB/";
    filename = filename + argv[1];
    cout << filename << endl;
    ReadFile(filename, n, cost, X, Y);

    allocateCudaMemory();

    defaultArr = new int[n];
    for(int i = 0; i < n; i++) 
        defaultArr[i] = i;
    
    makeInitialPopulation();

    
}

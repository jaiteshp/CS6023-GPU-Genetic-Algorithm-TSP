#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "TSPLIB_parser.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <ctime>

using namespace std;
#define dbg cout << __FILE__ << ":" << __LINE__ << ", " << endl

const int POP_SIZE = 100;
const int NUM_GEN = 1;
const float MUTATION_RATE = 0.05;
int NUM_MUTATIONS;
int m = POP_SIZE;
int n;
double **cost, **d_cost;
double *X, *Y, *d_X, *d_Y;
int *defaultArr;
int **initialPopulation;
int **pop1, **pop2, **ofsp;
float *rndm;
int RNDM_NUM_COUNT;

void allocateCudaMemory() {
    double **temp = (double**) malloc(sizeof(double*)*n);
    for(int i = 0; i < n; i++) {
        hipMalloc(&temp[i], sizeof(double)*n);
        hipMemcpy(&temp[i], cost[i], sizeof(double)*n, hipMemcpyHostToDevice);
    }

    hipMalloc(&d_cost, sizeof(double*)*n);
    hipMemcpy(&d_cost, temp, sizeof(double*)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_X, sizeof(double)*n);
    hipMemcpy(&d_X, X, sizeof(double)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_Y, sizeof(double)*n);
    hipMemcpy(&d_Y, Y, sizeof(double)*n, hipMemcpyHostToDevice);
    
    return;
}

void makeInitialPopulation() {
    hipMalloc(&rndm, sizeof(float)*RNDM_NUM_COUNT);
    int **cpop1, **cpop2, **cofsp;
    cpop1 = new int*[POP_SIZE];
    cpop2 = new int*[POP_SIZE];
    cofsp = new int*[POP_SIZE];
    hipMallocManaged(&initialPopulation, sizeof(int*)*POP_SIZE);
    hipMalloc(&pop1, sizeof(int*)*POP_SIZE);
    hipMalloc(&pop2, sizeof(int*)*POP_SIZE);
    hipMalloc(&ofsp, sizeof(int*)*POP_SIZE);
    for(int i = 0; i < POP_SIZE; i++) {
        hipMallocManaged(&initialPopulation[i], sizeof(int)*n);
        hipMalloc(&cpop1[i], sizeof(int)*n);
        hipMalloc(&cpop2[i], sizeof(int)*n);
        hipMalloc(&cofsp[i], sizeof(int)*n);
        random_shuffle(defaultArr, defaultArr+n);
        for(int j = 0; j < n; j++) initialPopulation[i][j] = defaultArr[j];
    }
    // hipMemcpy(initialPopulation, cinitialPopulation, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(pop1, cpop1, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(pop2, cpop2, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(ofsp, cofsp, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    // for(int i = 0; i < POP_SIZE; i++) {
    //     for(int j = 0; j < n; j++) {
    //         cout << initialPopulation[i][j] << ",";
    //     }
    //     cout << endl;
    // }
    return;    
}

__global__ void copyKernel(int n, int POP_SIZE, int **pop1, int **pop2) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= POP_SIZE) 
        return;
    
    for(int i = 0; i < n; i++) 
        pop1[id][i] = pop2[id][i];

    return;
}

__global__ void processKernel(int n, int POP_SIZE, int NUM_MUTATIONS, int **pop1, int **pop2, double **cost, double *X, double *Y, float *rndm) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= POP_SIZE) 
        return;
}

void generateRandomNumbers() {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);    
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned int)time(NULL));
    hiprandGenerateUniform(gen, rndm, RNDM_NUM_COUNT);
    hiprandDestroyGenerator(gen);
    hipDeviceSynchronize();
}

void runGA() {
    for(int genNum = 0; genNum < NUM_GEN; genNum++) {
        if(genNum == 0) 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, initialPopulation);        
        else 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, pop2);
        hipDeviceSynchronize();

        generateRandomNumbers();

        processKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, NUM_MUTATIONS, pop1, pop2, d_cost, d_X, d_Y, rndm);
        hipDeviceSynchronize();
    }
    return;
}

int main(int argc, char **argv) {
    string filename = "TSPLIB/";
    filename = filename + argv[1];
    cout << filename << endl;
    ReadFile(filename, n, cost, X, Y);

    NUM_MUTATIONS = n*MUTATION_RATE;
    RNDM_NUM_COUNT = POP_SIZE*(4 + 2*NUM_MUTATIONS);

    allocateCudaMemory();

    defaultArr = new int[n];
    for(int i = 0; i < n; i++) 
        defaultArr[i] = i;
    
    makeInitialPopulation();

    runGA();
}

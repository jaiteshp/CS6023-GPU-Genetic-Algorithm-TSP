#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "TSPLIB_parser.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <ctime>

using namespace std;
using std::cout; using std::endl;
using std::chrono::duration_cast;
using std::chrono::milliseconds;
using std::chrono::seconds;
using std::chrono::system_clock;

#define dbg cout << __FILE__ << ":" << __LINE__ << ", " << endl
// #define DBL_MAX 1.7976931348623158e+307

const int POP_SIZE = 7000;
const int NUM_GEN = 3000;
const float MUTATION_RATE = 0.001;
int NUM_MUTATIONS = 1;
int m = POP_SIZE;
int n;
double **d_cost1, **d_cost2;
double **cost, **d_cost;
double **ccost;
double *X, *Y, *d_X, *d_Y;
int *defaultArr;
int **initialPopulation;
int **pop1, **pop2, **ofsp;
float *rndm;
int RNDM_NUM_COUNT;

void allocateCudaMemory() {
    // double **temp = (double**) malloc(sizeof(double*)*n);
    double **temp = new double*[n];
    for(int i = 0; i < n; i++) {
        hipMalloc(&temp[i], sizeof(double)*n);
        hipMemcpy(&temp[i], cost[i], sizeof(double)*n, hipMemcpyHostToDevice);
    }

    hipMalloc(&d_cost, sizeof(double*)*n);
    hipMemcpy(&d_cost, temp, sizeof(double*)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_X, sizeof(double)*n);
    hipMemcpy(&d_X, X, sizeof(double)*n, hipMemcpyHostToDevice);

    hipMalloc(&d_Y, sizeof(double)*n);
    hipMemcpy(&d_Y, Y, sizeof(double)*n, hipMemcpyHostToDevice);
    
    return;
}

__global__ void printCostRow(int n, double *row) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id > 0)
        return;
    for(int i = 0; i < n; i++) {
        printf("%lf\t", row[i]);
    }
    printf("\n");
    return;
}

void makeInitialPopulation() {
    hipMalloc(&rndm, sizeof(float)*RNDM_NUM_COUNT);
    int **cpop1, **cpop2, **cofsp;
    // ccost = new double*[n];
    ccost = (double **) malloc(sizeof(double*)*n);
    cpop1 = new int*[POP_SIZE];
    cpop2 = new int*[POP_SIZE];
    cofsp = new int*[POP_SIZE];
    hipMallocManaged(&initialPopulation, sizeof(int*)*POP_SIZE);
    hipMallocManaged(&d_cost2, sizeof(double*)*n);
    hipMalloc(&d_cost1, sizeof(double*)*n);
    hipMalloc(&pop1, sizeof(int*)*POP_SIZE);
    hipMalloc(&pop2, sizeof(int*)*POP_SIZE);
    hipMalloc(&ofsp, sizeof(int*)*POP_SIZE);
    for(int i = 0; i < n; i++) {
        hipMalloc(&ccost[i], sizeof(double)*n);
        if(i == 0) {
            dbg;
            for(int j = 0; j < n; j++) {
                cout << cost[i][j] << "\t";
            }
            cout << endl;
            dbg;
        }
        hipMemcpy(&ccost[i], cost[i], sizeof(double)*n, hipMemcpyHostToDevice);
        hipMallocManaged(&d_cost2[i], sizeof(double)*n);
    }
    for(int i = 0; i < POP_SIZE; i++) {
        hipMallocManaged(&initialPopulation[i], sizeof(int)*n);
        hipMalloc(&cpop1[i], sizeof(int)*n);
        hipMalloc(&cpop2[i], sizeof(int)*n);
        hipMalloc(&cofsp[i], sizeof(int)*n);
        random_shuffle(defaultArr, defaultArr+n);
        for(int j = 0; j < n; j++) initialPopulation[i][j] = defaultArr[j];
    }
    // hipMemcpy(initialPopulation, cinitialPopulation, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_cost1, ccost, sizeof(double*)*n, hipMemcpyHostToDevice);
    hipMemcpy(pop1, cpop1, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(pop2, cpop2, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(ofsp, cofsp, sizeof(int*)*POP_SIZE, hipMemcpyHostToDevice);
    // for(int i = 0; i < POP_SIZE; i++) {
    //     for(int j = 0; j < n; j++) {
    //         cout << initialPopulation[i][j] << ",";
    //     }
    //     cout << endl;
    // }
    return;    
}

__global__ void copyKernel(int n, int POP_SIZE, int **pop1, int **pop2) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= POP_SIZE) 
        return;
    
    for(int i = 0; i < n; i++) 
        pop1[id][i] = pop2[id][i];

    return;
}

__device__ double computeFitness(int n, int **pop1, int row, double **cost) {
    double pathLength = 0.0;
    for(int i = 1; i < n; i++) {
        int u = pop1[row][i-1];
        int v = pop1[row][i];
        if(u < 0 || u >= n || v < 0 || v >= n) return pathLength;
        // printf("%d %d\n", u, v);
        pathLength = pathLength + cost[u][v];
    }
    pathLength += cost[pop1[row][n-1]][pop1[row][0]];
    return pathLength;
}

__device__ int argMaxFitness(int n, int **pop1, int low, int high, double **cost) {
    int idx = 0;
    double mn = 1.7976931348623158e+40;
    // printf("hi 106, %d %d\n", low, high);
    // return idx;
    for(int row = low; row < high; row++) {
        double fitness = computeFitness(n, pop1, row, cost);
        if(fitness < mn) {
            mn = fitness;
            idx = row;
        }
    }
    return idx;
}

__device__ int getAvlblIdx(int &idx, int n, int a, int b) {
    int res;
    if(idx < a || idx > b) {
        res = idx;
        idx++;
    } else {
        res = b+1;
        idx = b+2;
    }
    return res;
}

__device__ void mutateOffspring(int id, int n, int NUM_MUTATIONS, int **pop2, float *rndm) {
    int offset = id*(6+2*(NUM_MUTATIONS))+6;
    for(int mut = 0; mut < NUM_MUTATIONS; mut++) {
        int a, b;
        a = n*rndm[offset++];
        b = n*rndm[offset++];

        int temp = pop2[id][a];
        pop2[id][a] = pop2[id][b];
        pop2[id][b] = temp;
    }
    return;
}

__device__ void adjustRangeOrder(int &a, int &b) {
    if(a > b) {
        int temp = a;
        a = b;
        b = temp;
    } else if(a == b) {
        if(a == 0) b++;
        else a--;
    }
    return ;
}

__global__ void processKernel(int n, int POP_SIZE, int NUM_MUTATIONS, int **pop1, int **pop2, int **pres, double **cost, double *X, double *Y, float *rndm) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id >= POP_SIZE) 
        return;
    
    int parent1, parent2, low1, high1, low2, high2, a, b;
    int offset = id*(6+2*(NUM_MUTATIONS));
    for(int i = 0; i < 4; i++) 
        rndm[offset+i] = POP_SIZE*rndm[offset+i];
    low1 = rndm[offset+0];
    high1 = rndm[offset+1];
    low2 = rndm[offset+2];
    high2 = rndm[offset+3];
    adjustRangeOrder(low1, high1);
    adjustRangeOrder(low2, high2);

    //////////////////////////////////
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            double temp = cost[i][j];
            temp = temp+1.0;
            cost[i][j] = temp-1.0;
        }
    }

    
    /////////////////////////////////////

    parent1 = argMaxFitness(n, pop1, low1, high1, cost);
    parent2 = argMaxFitness(n, pop1, low2, high2, cost);
    

    a = n*rndm[offset+4];
    b = n*rndm[offset+5];
    adjustRangeOrder(a, b);

    for(int i = 0; i < n; i++) 
        pres[id][i] = 0;

   
    for(int i = a; i <= b; i++) {
        pop2[id][i] = pop1[parent1][i];
        pres[id][pop1[parent1][i]] = 1;
    }

    
    int avlblIdx = 0;
    for(int i = 0; i < n; i++) {
        int numToInsert = pop1[parent2][i];
        if(pres[id][numToInsert] == 0) {
            pres[id][numToInsert] = 1;
            pop2[id][getAvlblIdx(avlblIdx, n, a, b)] = numToInsert;
        }
    }


    mutateOffspring(id, n, NUM_MUTATIONS, pop2, rndm);
    if(id < 2) 
        printf("%d success %lf\n", id, computeFitness(n, pop2, id, cost));
    return;    
}

void generateRandomNumbers() {
    hiprandGenerator_t gen;    
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);      
    auto millisec_since_epoch = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned int) millisec_since_epoch);  
    // cout << "Time " << millisec_since_epoch << endl;
    hiprandGenerateUniform(gen, rndm, RNDM_NUM_COUNT);   
    hiprandDestroyGenerator(gen);    
    hipDeviceSynchronize();  
    
}

__global__ void printCost(int n, double **cost) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id > 0) return;
    printf("237, %d\n", n);
    int i, j;
    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            printf("%lf\t", cost[i][j]);
            // printf("yo\t");
        }
        printf("\n");
    }
    printf("hi\n");
    return;
}

__global__ void copyD_cost2ToD_cost1(int n, double **cost1, double **cost2) {
    int id = (blockIdx.x*blockDim.x)+threadIdx.x;
    if(id > 0)
        return;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cost1[i][j] = cost2[i][j];
        }
    }
    return;
}

void runGA() {
    for(int genNum = 0; genNum < NUM_GEN; genNum++) {
        cout << "#####################" << genNum << "#######################" << endl;
        if(genNum == 0) 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, initialPopulation);        
        else 
            copyKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, pop1, pop2);
        hipDeviceSynchronize();
        generateRandomNumbers();
        processKernel<<<ceil(POP_SIZE/(float) 1024), 1024>>>(n, POP_SIZE, NUM_MUTATIONS, pop1, pop2, ofsp, d_cost1, d_X, d_Y, rndm);
        hipDeviceSynchronize();
    }
    return;
}

void printCPUCost() {
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cout << (double) cost[i][j] << "\t";
        }
        cout << endl;
    }
    return;
}

void transposeCosts() {
    for(int i = 0; i < n; i++) {
        for(int j = n-1; j > i; j--) {
            cost[i][j] = cost[j][i];
        }
    }
    return;
}

void copyCostsTod_cost2(){
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            d_cost2[i][j] = cost[i][j];
        }
    }
}

int main(int argc, char **argv) {
    string filename = "TSPLIB/";
    filename = filename + argv[1];
    cout << filename << endl;
    ReadFile(filename, n, cost, X, Y);

    // NUM_MUTATIONS = n*MUTATION_RATE;
    // NUM_MUTATIONS = 1;
    RNDM_NUM_COUNT = POP_SIZE*(6 + 2*NUM_MUTATIONS);

    transposeCosts();

    allocateCudaMemory();

    defaultArr = new int[n];
    for(int i = 0; i < n; i++) 
        defaultArr[i] = i;
    
    makeInitialPopulation();
    copyCostsTod_cost2();
    copyD_cost2ToD_cost1<<<1,1>>>(n, d_cost1, d_cost2);

    dbg;
    runGA();
    dbg;
    // for(int i = 0; i < 1000; i++) generateRandomNumbers();

    // dbg;
    // printCost<<<1,1>>>(n, d_cost1);
    // hipDeviceSynchronize();
    // dbg;
    // printCPUCost();
    // dbg;
    // printCostRow<<<1,1>>>(n, ccost[0]);
    // hipDeviceSynchronize();
    // dbg;
    return 0;
}
